#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <ctime>
#include <iomanip>

#include "hip/hip_runtime.h"
#include ""
#include "bitmap_image.hpp" // clase para leer im�genes BMP descargada de: http://www.partow.net/programming/bitmap/index.html
#include "../common/lectura_datos.h"  // para lectura de im�genes y vol�menes 3D
#include "regression_line.h"

#include <stdio.h>

// seleccionar uno de estos dos defines para ejecutar la suma mediante sumas at�micas o mediante reducci�n
#define _ATOMIC_SUM 1  
//#define _REDUCTION_SUM 1 

// seleccionar uno de estos dos defines para hacer las transferencias de memoria utilizando memoria host pinned o memoria host normal (_PAGEABLE)
#define _PINNED 1 // ESTE ES EL QUE DA MEJORES TIEMPOS
//#define _PAGEABLE 1

// N�mero m�ximo de threads por bloque es 1024 para compute capability > 2.0
#define TPB 128 // threads per block mejor para Tesla K40c: 128
//#define TPB 64 // threads per block mejor para GTX 850m: 64

// seleccionar seg�n corresponda en base al valor de TPB
#define TPB_POTENCIA_2 1
//#define TPB_NO_POTENCIA_2 1



/**
@brief calcula el m�ximo y el m�nimo utilizando sentencias if
*/
__device__ void max_min(const unsigned char a, const unsigned char b, const unsigned char c, const unsigned char d,
	const unsigned char e, const unsigned char f, const unsigned char g, const unsigned char h,
	const unsigned char i, const unsigned char j, const unsigned char k, const unsigned char l,
	const unsigned char m, const unsigned char n, const unsigned char o, const unsigned char p,
	const unsigned char a2, const unsigned char b2, const unsigned char c2, const unsigned char d2,
	const unsigned char e2, const unsigned char f2, const unsigned char g2, const unsigned char h2,
	const unsigned char i2, const unsigned char j2, const unsigned char k2, const unsigned char l2,
	const unsigned char m2, const unsigned char n2, const unsigned char o2, const unsigned char p2,
	unsigned char& maxv, unsigned char& minv) {
	if ((a >= b) && (a >= c) && (a >= d) && (a >= e) && (a >= f) && (a >= g) && (a >= h) &&
		(a >= i) && (a >= j) && (a >= k) && (a >= l) && (a >= m) && (a >= n) && (a >= o) && (a >= p)) maxv = a;
	else if ((b >= a) && (b >= c) && (b >= d) && (b >= e) && (b >= f) && (b >= g) && (b >= h) &&
		(b >= i) && (b >= j) && (b >= k) && (b >= l) && (b >= m) && (b >= n) && (b >= o) && (b >= p)) maxv = b;
	else if ((c >= a) && (c >= b) && (c >= d) && (c >= e) && (c >= f) && (c >= g) && (c >= h) &&
		(c >= i) && (c >= j) && (c >= k) && (c >= l) && (c >= m) && (c >= n) && (c >= o) && (c >= p)) maxv = c;
	else if ((d >= a) && (d >= b) && (d >= c) && (d >= e) && (d >= f) && (d >= g) && (d >= h) &&
		(d >= i) && (d >= j) && (d >= k) && (d >= l) && (d >= m) && (d >= n) && (d >= o) && (d >= p)) maxv = d;
	else if ((e >= a) && (e >= b) && (e >= c) && (e >= d) && (e >= f) && (e >= g) && (e >= h) &&
		(e >= i) && (e >= j) && (e >= k) && (e >= l) && (e >= m) && (e >= n) && (e >= o) && (e >= p)) maxv = e;
	else if ((f >= a) && (f >= b) && (f >= c) && (f >= d) && (f >= e) && (f >= g) && (f >= h) &&
		(f >= i) && (f >= j) && (f >= k) && (f >= l) && (f >= m) && (f >= n) && (f >= o) && (f >= p)) maxv = f;
	else if ((g >= a) && (g >= b) && (g >= c) && (g >= d) && (g >= e) && (g >= f) && (g >= h) &&
		(g >= i) && (g >= j) && (g >= k) && (g >= l) && (g >= m) && (g >= n) && (g >= o) && (g >= p)) maxv = g;
	else if ((h >= a) && (h >= b) && (h >= c) && (h >= d) && (h >= e) && (h >= f) && (h >= g) &&
		(h >= i) && (h >= j) && (h >= k) && (h >= l) && (h >= m) && (h >= n) && (h >= o) && (h >= p)) maxv = h;
	else if ((i >= a) && (i >= b) && (i >= c) && (i >= d) && (i >= e) && (i >= f) && (i >= g) &&
		(i >= h) && (i >= j) && (i >= k) && (i >= l) && (i >= m) && (i >= n) && (i >= o) && (i >= p)) maxv = i;
	else if ((j >= a) && (j >= b) && (j >= c) && (j >= d) && (j >= e) && (j >= f) && (j >= g) &&
		(j >= h) && (j >= i) && (j >= k) && (j >= l) && (j >= m) && (j >= n) && (j >= o) && (j >= p)) maxv = j;
	else if ((k >= a) && (k >= b) && (k >= c) && (k >= d) && (k >= e) && (k >= f) && (k >= g) &&
		(k >= h) && (k >= i) && (k >= j) && (k >= l) && (k >= m) && (k >= n) && (k >= o) && (k >= p)) maxv = k;
	else if ((l >= a) && (l >= b) && (l >= c) && (l >= d) && (l >= e) && (l >= f) && (l >= g) &&
		(l >= h) && (l >= i) && (l >= j) && (l >= k) && (l >= m) && (l >= n) && (l >= o) && (l >= p)) maxv = l;
	else if ((m >= a) && (m >= b) && (m >= c) && (m >= d) && (m >= e) && (m >= f) && (m >= g) &&
		(m >= h) && (m >= i) && (m >= j) && (m >= k) && (m >= l) && (m >= n) && (m >= o) && (m >= p)) maxv = m;
	else if ((n >= a) && (n >= b) && (n >= c) && (n >= d) && (n >= e) && (n >= f) && (n >= g) &&
		(n >= h) && (n >= i) && (n >= j) && (n >= k) && (n >= l) && (n >= m) && (n >= o) && (n >= p)) maxv = n;
	else if ((o >= a) && (o >= b) && (o >= c) && (o >= d) && (o >= e) && (o >= f) && (o >= g) &&
		(o >= h) && (o >= i) && (o >= j) && (o >= k) && (o >= l) && (o >= m) && (o >= n) && (o >= p)) maxv = o;
	else maxv = p;

	if ((a2 <= b2) && (a2 <= c2) && (a2 <= d2) && (a2 <= e2) && (a2 <= f2) && (a2 <= g2) && (a2 <= h2) &&
		(a2 <= i2) && (a2 <= j2) && (a2 <= k2) && (a2 <= l2) && (a2 <= m2) && (a2 <= n2) && (a2 <= o2) && (a2 <= p2)) minv = a2;
	else if ((b2 <= a2) && (b2 <= c2) && (b2 <= d2) && (b2 <= e2) && (b2 <= f2) && (b2 <= g2) && (b2 <= h2) &&
		(b2 <= i2) && (b2 <= j2) && (b2 <= k2) && (b2 <= l2) && (b2 <= m2) && (b2 <= n2) && (b2 <= o2) && (b2 <= p2)) minv = b2;
	else if ((c2 <= a2) && (c2 <= b2) && (c2 <= d2) && (c2 <= e2) && (c2 <= f2) && (c2 <= g2) && (c2 <= h2) &&
		(c2 <= i2) && (c2 <= j2) && (c2 <= k2) && (c2 <= l2) && (c2 <= m2) && (c2 <= n2) && (c2 <= o2) && (c2 <= p2)) minv = c2;
	else if ((d2 <= a2) && (d2 <= b2) && (d2 <= c2) && (d2 <= e2) && (d2 <= f2) && (d2 <= g2) && (d2 <= h2) &&
		(d2 <= i2) && (d2 <= j2) && (d2 <= k2) && (d2 <= l2) && (d2 <= m2) && (d2 <= n2) && (d2 <= o2) && (d2 <= p2)) minv = d2;
	else if ((e2 <= a2) && (e2 <= b2) && (e2 <= c2) && (e2 <= d2) && (e2 <= f2) && (e2 <= g2) && (e2 <= h2) &&
		(e2 <= i2) && (e2 <= j2) && (e2 <= k2) && (e2 <= l2) && (e2 <= m2) && (e2 <= n2) && (e2 <= o2) && (e2 <= p2)) minv = e2;
	else if ((f2 <= a2) && (f2 <= b2) && (f2 <= c2) && (f2 <= d2) && (f2 <= e2) && (f2 <= g2) && (f2 <= h2) &&
		(f2 <= i2) && (f2 <= j2) && (f2 <= k2) && (f2 <= l2) && (f2 <= m2) && (f2 <= n2) && (f2 <= o2) && (f2 <= p2)) minv = f2;
	else if ((g2 <= a2) && (g2 <= b2) && (g2 <= c2) && (g2 <= d2) && (g2 <= e2) && (g2 <= f2) && (g2 <= h2) &&
		(g2 <= i2) && (g2 <= j2) && (g2 <= k2) && (g2 <= l2) && (g2 <= m2) && (g2 <= n2) && (g2 <= o2) && (g2 <= p2)) minv = g2;
	else if ((h2 <= a2) && (h2 <= b2) && (h2 <= c2) && (h2 <= d2) && (h2 <= e2) && (h2 <= f2) && (h2 <= g2) &&
		(h2 <= i2) && (h2 <= j2) && (h2 <= k2) && (h2 <= l2) && (h2 <= m2) && (h2 <= n2) && (h2 <= o2) && (h2 <= p2)) minv = h2;
	else if ((i2 <= a2) && (i2 <= b2) && (i2 <= c2) && (i2 <= d2) && (i2 <= e2) && (i2 <= f2) && (i2 <= g2) &&
		(i2 <= h2) && (i2 <= j2) && (i2 <= k2) && (i2 <= l2) && (i2 <= m2) && (i2 <= n2) && (i2 <= o2) && (i2 <= p2)) minv = i2;
	else if ((j2 <= a2) && (j2 <= b2) && (j2 <= c2) && (j2 <= d2) && (j2 <= e2) && (j2 <= f2) && (j2 <= g2) &&
		(j2 <= h2) && (j2 <= i2) && (j2 <= k2) && (j2 <= l2) && (j2 <= m2) && (j2 <= n2) && (j2 <= o2) && (j2 <= p2)) minv = j2;
	else if ((k2 <= a2) && (k2 <= b2) && (k2 <= c2) && (k2 <= d2) && (k2 <= e2) && (k2 <= f2) && (k2 <= g2) &&
		(k2 <= h2) && (k2 <= i2) && (k2 <= j2) && (k2 <= l2) && (k2 <= m2) && (k2 <= n2) && (k2 <= o2) && (k2 <= p2)) minv = k2;
	else if ((l2 <= a2) && (l2 <= b2) && (l2 <= c2) && (l2 <= d2) && (l2 <= e2) && (l2 <= f2) && (l2 <= g2) &&
		(l2 <= h2) && (l2 <= i2) && (l2 <= j2) && (l2 <= k2) && (l2 <= m2) && (l2 <= n2) && (l2 <= o2) && (l2 <= p2)) minv = l2;
	else if ((m2 <= a2) && (m2 <= b2) && (m2 <= c2) && (m2 <= d2) && (m2 <= e2) && (m2 <= f2) && (m2 <= g2) &&
		(m2 <= h2) && (m2 <= i2) && (m2 <= j2) && (m2 <= k2) && (m2 <= l2) && (m2 <= n2) && (m2 <= o2) && (m2 <= p2)) minv = m2;
	else if ((n2 <= a2) && (n2 <= b2) && (n2 <= c2) && (n2 <= d2) && (n2 <= e2) && (n2 <= f2) && (n2 <= g2) &&
		(n2 <= h2) && (n2 <= i2) && (n2 <= j2) && (n2 <= k2) && (n2 <= l2) && (n2 <= m2) && (n2 <= o2) && (n2 <= p2)) minv = n2;
	else if ((o2 <= a2) && (o2 <= b2) && (o2 <= c2) && (o2 <= d2) && (o2 <= e2) && (o2 <= f2) && (o2 <= g2) &&
		(o2 <= h2) && (o2 <= i2) && (o2 <= j2) && (o2 <= k2) && (o2 <= l2) && (o2 <= m2) && (o2 <= n2) && (o2 <= p2)) minv = o2;
	else minv = p2;

}


/**
@brief Kernel CUDA que implementa el DBC 3D para el tama�o de grid m�s peque�o, de manera que Imin se inicializa con los valores obtenidos
	   de Imax, evitando la copia inicial desde Imax a Imin
@param bits_M [in] 2^bits_M = M
@param sm [in] mitad del lado del box (lado del box = s)
@param sp [in] altura del box en la dimensi�n z
@param bits_s [i] 2^bits_s = s (s: lado del box)
*/

__global__ void DBCKernel_inicial(unsigned char* Imax, unsigned char* Imin, const int M, const unsigned char bits_M, const int G, unsigned int* Nr,
	const unsigned int sm, const unsigned int sp, const unsigned char bits_s,
	const unsigned char bits_TPB)
{
	#ifdef _REDUCTION_SUM
	__shared__ unsigned int SharedData[TPB]; // stores box-counting of each thread for using it in reduction sum
											 // este TPB tiene que ser fijo y siempre es mayor o igual a vTPB
	#endif

	

	register unsigned long long int tid = threadIdx.x; // identificaci�n del thread dentro del bloque
	register unsigned long long int idx = (blockIdx.x << bits_TPB /** vTPB*/) + tid; // identificaci�n del thread global

	/* ATENCI�N!!!: esta comprobaci�n solo hace falta si TPB no es potencia de 2 */
	/* si el TPB elegido finalmente es potencia de 2, entonces quitar este if para mejorar la ejecuci�n */
#ifdef TPB_NO_POTENCIA_2
	if (idx >= ((M >> 1) << ((bits_M - 1) + (bits_M - 1) + (bits_M - 1))))
		return;
#endif
	register unsigned int l = idx >> ((bits_M - bits_s) + (bits_M - bits_s) + (bits_M - bits_s)); // l index: idx/((m/s)*(m/s)*(m/s))
	register unsigned int offsetl = (idx & (((M >> bits_s) << ((bits_M - bits_s) + (bits_M - bits_s))) - 1)); // idx mod ((m/s)*(m/s)*(m/s)): offset inside 3D matrix l

	register unsigned int k = offsetl >> ((bits_M - bits_s) + (bits_M - bits_s)); // k index: offsetl / ((m/s)*(m/s))
	register unsigned int offset = (offsetl & (((M >> bits_s) << (bits_M - bits_s)) - 1)); // (offsetl mod ((m/s)*(m/s))), offset inside k slice

	register unsigned long long int i = offset >> (bits_M - bits_s); // i index: offset / (m/s)
	register unsigned long long int j = offset & ((M >> bits_s) - 1); // j index: offset mod (m/s)


	const register unsigned int jbs = j << bits_s;
	const register unsigned int ibsbM = (i << bits_s) << bits_M;
	const register unsigned int kbsbMM = ((k << bits_s) << bits_M) << bits_M;
	const register unsigned int lbsbMMM = (((l << bits_s) << bits_M) << bits_M) << bits_M;
	
	const register unsigned int jbssm = (j << bits_s) + sm;
	const register unsigned int ibssmbM = ((i << bits_s) + sm) << bits_M;
	const register unsigned int kbssmbMM = (((k << bits_s) + sm) << bits_M) << bits_M;
	const register unsigned int lbssmbMMM = ((((l << bits_s) + sm) << bits_M) << bits_M) << bits_M;

	
	register unsigned char valmax;
	register unsigned char valmin;
	max_min( Imax[lbsbMMM + kbsbMM + ibsbM + jbs], Imax[lbsbMMM + kbsbMM + ibsbM + jbssm], 
				Imax[lbsbMMM + kbsbMM + ibssmbM + jbs], Imax[lbsbMMM + kbsbMM + ibssmbM + jbssm],
				Imax[lbsbMMM + kbssmbMM + ibsbM + jbs], Imax[lbsbMMM + kbssmbMM + ibsbM + jbssm], 
				Imax[lbsbMMM + kbssmbMM + ibssmbM + jbs], Imax[lbsbMMM + kbssmbMM + ibssmbM + jbssm],
				Imax[lbssmbMMM + kbsbMM + ibsbM + jbs], Imax[lbssmbMMM + kbsbMM + ibsbM + jbssm], 
				Imax[lbssmbMMM + kbsbMM + ibssmbM + jbs], Imax[lbssmbMMM + kbsbMM + ibssmbM + jbssm],
				Imax[lbssmbMMM + kbssmbMM + ibsbM + jbs], Imax[lbssmbMMM + kbssmbMM + ibsbM + jbssm], 
				Imax[lbssmbMMM + kbssmbMM + ibssmbM + jbs], Imax[lbssmbMMM + kbssmbMM + ibssmbM + jbssm],

				Imax[lbsbMMM + kbsbMM + ibsbM + jbs], Imax[lbsbMMM + kbsbMM + ibsbM + jbssm],
				Imax[lbsbMMM + kbsbMM + ibssmbM + jbs], Imax[lbsbMMM + kbsbMM + ibssmbM + jbssm],
				Imax[lbsbMMM + kbssmbMM + ibsbM + jbs], Imax[lbsbMMM + kbssmbMM + ibsbM + jbssm],
				Imax[lbsbMMM + kbssmbMM + ibssmbM + jbs], Imax[lbsbMMM + kbssmbMM + ibssmbM + jbssm],
				Imax[lbssmbMMM + kbsbMM + ibsbM + jbs], Imax[lbssmbMMM + kbsbMM + ibsbM + jbssm],
				Imax[lbssmbMMM + kbsbMM + ibssmbM + jbs], Imax[lbssmbMMM + kbsbMM + ibssmbM + jbssm],
				Imax[lbssmbMMM + kbssmbMM + ibsbM + jbs], Imax[lbssmbMMM + kbssmbMM + ibsbM + jbssm],
				Imax[lbssmbMMM + kbssmbMM + ibssmbM + jbs], Imax[lbssmbMMM + kbssmbMM + ibssmbM + jbssm],
				valmax, valmin);


	Imax[lbsbMMM + kbsbMM + ibsbM + jbs] = valmax;
	Imin[lbsbMMM + kbsbMM + ibsbM + jbs] = valmin;

	/* usando sumas at�micas */
#ifdef _ATOMIC_SUM
	float invsp = 1.0 / sp;
	atomicAdd(Nr, ceilf((float) /*valmax[15]*//*maxv*/valmax * invsp) - ceilf((float) /*valmin[15]*//*minv*/valmin * invsp) + 1);
#endif
	/* fin usando sumas at�micas*/

	/* usando sumas por reducci�n */
#ifdef _REDUCTION_SUM
	float invsp = 1.0 / sp;
	SharedData[tid] = ceilf((float) /*valmax[15]*/maxv * invsp) - ceilf((float) /*valmin[15]*/minv * invsp) + 1;
	__syncthreads();
	for (unsigned int ss = (vTPB >> 1); ss > 0; ss >>= 1) {
		if (tid < ss) SharedData[tid] += SharedData[tid + ss];
		__syncthreads();
	}

	if (tid == 0) atomicAdd(Nr, *SharedData); // result of reduction sum is returned
#endif
/* fin usando sumas por reducci�n */
}


/**
@brief Kernel CUDA que implementa el DBC 4D
@param bits_M [in] 2^bits_M = M
@param sm [in] mitad del lado del box (lado del box = s)
@param sp [in] altura del box en la dimensi�n z
@param bits_s [i] 2^bits_s = s (s: lado del box)
*/
__global__ void DBCKernel(unsigned char* Imax, unsigned char* Imin, const int M,
	const unsigned char bits_M, const int G, unsigned int* Nr,
	const unsigned int sm, const unsigned int sp, const unsigned char bits_s,
	const unsigned char bits_TPB)
{
#ifdef _REDUCTION_SUM
	__shared__ unsigned int SharedData[TPB]; // stores box-counting of each thread for using it in reduction sum
											 // este TPB tiene que ser fijo y siempre es mayor o igual a vTPB
#endif

	register unsigned long long int tid = threadIdx.x; // identificaci�n del thread dentro del bloque
	register unsigned long long int idx = (blockIdx.x << bits_TPB /** vTPB*/) + tid; // identificaci�n del thread global

	/* ATENCI�N!!!: esta comprobaci�n solo hace falta si TPB no es potencia de 2 */
	/* si el TPB elegido finalmente es potencia de 2, entonces quitar este if para mejorar la ejecuci�n */
#ifdef TPB_NO_POTENCIA_2
	if (idx >= ((M >> 1) << ((bits_M - 1) + (bits_M - 1) + (bits_M - 1))))
		return;
#endif
	register unsigned int l = idx >> ((bits_M - bits_s) + (bits_M - bits_s) + (bits_M - bits_s)); // l index: idx/((m/s)*(m/s)*(m/s))
	register unsigned int offsetl = (idx & (((M >> bits_s) << ((bits_M - bits_s) + (bits_M - bits_s))) - 1)); // idx mod ((m/s)*(m/s)*(m/s)): offset inside 3D matrix l

	register unsigned int k = offsetl >> ((bits_M - bits_s) + (bits_M - bits_s)); // k index: offsetl / ((m/s)*(m/s))
	register unsigned int offset = (offsetl & (((M >> bits_s) << (bits_M - bits_s)) - 1)); // (offsetl mod ((m/s)*(m/s))), offset inside k slice

	register unsigned long long int i = offset >> (bits_M - bits_s); // i index: offset / (m/s)
	register unsigned long long int j = offset & ((M >> bits_s) - 1); // j index: offset mod (m/s)


	const register unsigned int jbs = j << bits_s;
	const register unsigned int ibsbM = (i << bits_s) << bits_M;
	const register unsigned int kbsbMM = ((k << bits_s) << bits_M) << bits_M;
	const register unsigned int lbsbMMM = (((l << bits_s) << bits_M) << bits_M) << bits_M;

	const register unsigned int jbssm = (j << bits_s) + sm;
	const register unsigned int ibssmbM = ((i << bits_s) + sm) << bits_M;
	const register unsigned int kbssmbMM = (((k << bits_s) + sm) << bits_M) << bits_M;
	const register unsigned int lbssmbMMM = ((((l << bits_s) + sm) << bits_M) << bits_M) << bits_M;


	register unsigned char valmax;
	register unsigned char valmin;
	max_min(Imax[lbsbMMM + kbsbMM + ibsbM + jbs], Imax[lbsbMMM + kbsbMM + ibsbM + jbssm],
		Imax[lbsbMMM + kbsbMM + ibssmbM + jbs], Imax[lbsbMMM + kbsbMM + ibssmbM + jbssm],
		Imax[lbsbMMM + kbssmbMM + ibsbM + jbs], Imax[lbsbMMM + kbssmbMM + ibsbM + jbssm],
		Imax[lbsbMMM + kbssmbMM + ibssmbM + jbs], Imax[lbsbMMM + kbssmbMM + ibssmbM + jbssm],
		Imax[lbssmbMMM + kbsbMM + ibsbM + jbs], Imax[lbssmbMMM + kbsbMM + ibsbM + jbssm],
		Imax[lbssmbMMM + kbsbMM + ibssmbM + jbs], Imax[lbssmbMMM + kbsbMM + ibssmbM + jbssm],
		Imax[lbssmbMMM + kbssmbMM + ibsbM + jbs], Imax[lbssmbMMM + kbssmbMM + ibsbM + jbssm],
		Imax[lbssmbMMM + kbssmbMM + ibssmbM + jbs], Imax[lbssmbMMM + kbssmbMM + ibssmbM + jbssm],

		Imin[lbsbMMM + kbsbMM + ibsbM + jbs], Imin[lbsbMMM + kbsbMM + ibsbM + jbssm],
		Imin[lbsbMMM + kbsbMM + ibssmbM + jbs], Imin[lbsbMMM + kbsbMM + ibssmbM + jbssm],
		Imin[lbsbMMM + kbssmbMM + ibsbM + jbs], Imin[lbsbMMM + kbssmbMM + ibsbM + jbssm],
		Imin[lbsbMMM + kbssmbMM + ibssmbM + jbs], Imin[lbsbMMM + kbssmbMM + ibssmbM + jbssm],
		Imin[lbssmbMMM + kbsbMM + ibsbM + jbs], Imin[lbssmbMMM + kbsbMM + ibsbM + jbssm],
		Imin[lbssmbMMM + kbsbMM + ibssmbM + jbs], Imin[lbssmbMMM + kbsbMM + ibssmbM + jbssm],
		Imin[lbssmbMMM + kbssmbMM + ibsbM + jbs], Imin[lbssmbMMM + kbssmbMM + ibsbM + jbssm],
		Imin[lbssmbMMM + kbssmbMM + ibssmbM + jbs], Imin[lbssmbMMM + kbssmbMM + ibssmbM + jbssm],
		valmax, valmin);

	Imax[lbsbMMM + kbsbMM + ibsbM + jbs] = valmax;
	Imin[lbsbMMM + kbsbMM + ibsbM + jbs] = valmin;

	/* usando sumas at�micas */
#ifdef _ATOMIC_SUM
	float invsp = 1.0 / sp;
	atomicAdd(Nr, ceilf((float) /*valmax[15]*//*maxv*/valmax * invsp) - ceilf((float) /*valmin[15]*//*minv*/valmin * invsp) + 1);
#endif
	/* fin usando sumas at�micas*/

	/* usando sumas por reducci�n */
#ifdef _REDUCTION_SUM
	float invsp = 1.0 / sp;
	SharedData[tid] = ceilf((float) /*valmax[15]*/maxv * invsp) - ceilf((float) /*valmin[15]*/minv * invsp) + 1;
	__syncthreads();
	for (unsigned int ss = (vTPB >> 1); ss > 0; ss >>= 1) {
		if (tid < ss) SharedData[tid] += SharedData[tid + ss];
		__syncthreads();
	}

	if (tid == 0) atomicAdd(Nr, *SharedData); // result of reduction sum is returned
#endif
/* fin usando sumas por reducci�n */
}

/**
@brief Helper function for using CUDA to compute DBC algorithm
*/

hipError_t DBCWithCuda(const unsigned char* Imax, const int Numr,
	const int M, const unsigned char bits_M, const int G, unsigned int* Nr)
{
	unsigned char* dev_Imax = 0;
	unsigned char* dev_Imin = 0;
	unsigned int* dev_Nr = 0;
	hipError_t cudaStatus;
	hipEvent_t start, stop;
	float time_transfers, total_time_kernels;
	float time_kernel;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_Imax, M * M * M * M * sizeof(unsigned char));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_Imin, M * M * M * M * sizeof(unsigned char));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_Nr, Numr * sizeof(unsigned int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}


	total_time_kernels = 0;
	time_transfers = 0;
	for (int rep = 0; rep < 10; rep++) { // repeticiones para obtener tiempos mayores para comparar

		// Launch the kernel on the GPU
		unsigned int num_box;
		unsigned int tam_grid;
		dim3 grid, block(TPB, 1, 1);

		unsigned int sp;
		unsigned int s = 2;
		unsigned int sm;
		unsigned int size = M;
		unsigned char Nri = 0;
		int tpb;
		unsigned char b_tpb = log(TPB) / log(2);

		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipEventRecord(start, 0);
		// Imax se copia de host a device. Imin no hace falta copiarlo ya que se genera a partir de Imax en la primera llamada al kernel para el grid m�s peque�o
		cudaStatus = hipMemcpy(dev_Imax, Imax, M * M * M * M * sizeof(unsigned char), hipMemcpyHostToDevice);
		cudaStatus = hipMemset(dev_Nr, 0, Numr * sizeof(unsigned int));
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&time_kernel, start, stop);
		time_transfers += time_kernel;

		// primera llamada se hace especial para generar Imin a partir de los valores iniciales de Imax y asi ahorrar la copia
		// inicial desde Imax a Imin
		sm = s >> 1; // la mitad de s
		sp = ceilf(((float)(G << (Nri + 1))) / (float)M); // tamaño del voxel en la direccion z
		//tam_grid = ceilf(((M * M * M * M) >> 4) / (float)TPB); // M/2 * M/2 * M/2 * M/2 = (tam_grid * TPB)
		grid.x = ceilf(((M * M * M * M) / (s * s * s * s)) / (float)TPB);
		grid.y = 1; grid.z = 1;

		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipEventRecord(start, 0);
		DBCKernel_inicial<<<grid, block>>>(dev_Imax, dev_Imin, M, bits_M, G, &dev_Nr[Nri], sm, sp, Nri + 1, b_tpb);
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&time_kernel, start, stop);
		total_time_kernels += time_kernel;
		cudaStatus = hipGetLastError();
		

		Nri++;
		s <<= 1;
		size >>= 1;

		while (size > 2) {
			sm = s >> 1; // la mitad de s
			sp = ceilf(((float)(G << (Nri + 1))) / (float)M); // tama�o del voxel en la direcci�n z
			num_box = (M * M * M * M) / (s * s * s * s);
			if (num_box >= TPB) {
				grid.x = ceilf(num_box / (float)TPB); // M/s * M/s * M/s * M/s= (tam_grid * TPB)
				grid.y = 1; grid.z = 1;
				//tpb = TPB;
			}
			else {
				grid.x = 1; grid.y = 1; grid.z = 1;
				block.x = num_box; block.y = 1; block.z = 1;
				//tpb = num_box;
				b_tpb = log(num_box) / log(2);
			}

			hipEventCreate(&start);
			hipEventCreate(&stop);
			hipEventRecord(start, 0);
			DBCKernel<<<grid, block>>>(dev_Imax, dev_Imin, M, bits_M, G, &dev_Nr[Nri], sm, sp, Nri + 1, b_tpb);
			hipEventRecord(stop, 0);
			hipEventSynchronize(stop);
			hipEventElapsedTime(&time_kernel, start, stop);
			total_time_kernels += time_kernel;

			Nri++;
			s <<= 1;
			size >>= 1;
		}

		hipDeviceSynchronize();

		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipEventRecord(start, 0);
		cudaStatus = hipMemcpy(Nr, dev_Nr, Numr * sizeof(unsigned int), hipMemcpyDeviceToHost);
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&time_kernel, start, stop);
		time_transfers += time_kernel;

	} // fin de repeticiones

	printf("Tiempo kernels:  %3.1f ms \n", total_time_kernels);
	printf("Tiempo transferencias:  %3.1f ms \n", time_transfers);

Error:
	hipFree(dev_Imax);
	hipFree(dev_Imin);
	hipFree(dev_Nr);

	return cudaStatus;
}



void generar_volumen4D(unsigned char** I, int M) {
	int total_elements = M * M * M * M;
	*I = (unsigned char*)malloc(total_elements * sizeof(unsigned char));
	for (int i = 0; i < total_elements; ++i) {
		(*I)[i] = rand() % 256; // Números aleatorios entre 0 y 255
		//(*I)[i] = 255;
		//(*I)[i] = i % 255;
	}
}

/**
@brief Funci�n main
*/
int main()
{	
	// Array de nombres de ficheros de imagen
	char* filenames[] = {
		"..\\imagenes\\image4D_8.vox", "..\\imagenes\\image4D_16.vox",
		"..\\imagenes\\image4D_32.vox", "..\\imagenes\\image4D_64.vox",
		"..\\imagenes\\image4D_128.vox"
	};
	char* filenames_linux[] = {
		"../imagenes/image4D_8.vox", "../imagenes/image4D_16.vox",
		"../imagenes/image4D_32.vox", "../imagenes/image4D_64.vox",
		"../imagenes/image4D_128.vox"
	};
	int numFiles = 5;


	// leer volumen VOX 

	hipError_t cudaStatus;
	
	int G = 256; // n�mero de niveles de gris en el volumen

	int Numr;
	unsigned int* Nr;

	int s[6] = { 2, 4, 8, 16, 32, 64 };

	for (int fileIndex = 0; fileIndex < numFiles; ++fileIndex) {
		unsigned char* I = NULL;
		int M;

		std::cout << "Imagen  " << filenames[fileIndex] << std::endl;

#ifdef _WIN32
		leer_volumen4D_vox(filenames[fileIndex], &I, &M);
#elif __linux__
		leer_volumen4D_vox(filenames_linux[fileIndex], &I, &M);
#endif

		Numr = log(M) / log(2) - 1;
		const unsigned char bits_M = Numr + 1; // 2^bits_M = M

		// copia la imagen I en los dos buffers que contienen el maximo y el minimo para cada tama�o s 
#ifdef _PINNED
		unsigned char* Imax;
		cudaStatus = hipHostMalloc((void**)&Imax, sizeof(unsigned char) * M * M * M * M);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "CUDA Runtime Error: %s\n",
				hipGetErrorString(cudaStatus));
			return 1;
		}
		cudaStatus = hipHostMalloc((void**)&Nr, sizeof(unsigned int) * Numr);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "CUDA Runtime Error: %s\n",
				hipGetErrorString(cudaStatus));
			return 1;
	}
#elif _PAGEABLE
		unsigned char* Imax = new unsigned char[M * M * M * M];
		Nr = new unsigned int[Numr];
#endif

		memcpy(Imax, I, sizeof(unsigned char) * M * M * M * M);
		for (int i = 0;i < Numr; i++) Nr[i] = 0;

		//std::cout << "Ejecutando DBC 4D CUDA" << std::endl;
		cudaStatus = DBCWithCuda(Imax, Numr, M, bits_M, G, Nr);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "DBCWithCuda failed!");
			return 1;
		}

		// visualizamos los resultados 
		for (int i = 0; i < Numr; i++) {
			std::cout << "s: " << (2 << i) << " -- Nr: " << Nr[i] << std::endl;
		}


		// hipDeviceReset must be called before exiting in order for profiling and
		// tracing tools such as Nsight and Visual Profiler to show complete traces.
		cudaStatus = hipDeviceReset();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceReset failed!");
			return 1;
		}

#ifdef _PINNED
		hipHostFree(Imax);
		hipHostFree(Nr);
#elif _PAGEABLE
		delete Imax;
		delete Nr;
#endif

		free(I); // libera el volumen
		
	}

	return 0;


}
